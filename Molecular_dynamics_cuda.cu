﻿#include <iostream>
#include <random>
#include <cmath>
#include <chrono>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <cassert>
#define blockSize (64)
#define mu1 (1)
#define mu1_bath (1)
#define mu0 (0)
#define beta0 (1)
#define dt (0.05)              // 時間刻み幅
#define dt_1 (1.0 / dt)
#define dt2 (dt * dt)
#define gamma0 (0.2)          // Langevin熱浴のgamma係数
#define gamma_t (gamma0 * dt)   
#define statistics_interval (1)

#ifdef _WIN32
    #define SLASH "\\"
#else
    #define SLASH "/"
#endif
using namespace std;

__global__ 
void setCurand(unsigned long long seed, hiprandState *state){
    unsigned int i_global = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, i_global, 0, &state[i_global]);
}
__global__
void qinit(double *q1, double *q0, const int size_)
{
    const int thread_id = threadIdx.x+blockDim.x*blockIdx.x;
    const int i = thread_id;
    if(i < size_) {
        q0[i] = q1[i] = 0.0;
    }
}
__global__
void statistics_reset_d(double* fluxC, const int model_size) {
    const int thread_id = threadIdx.x+blockDim.x*blockIdx.x;
    const int i = thread_id + 1;
    if(i < model_size)
        fluxC[i] = 3.5;
}

__global__
void statistics_reset_test(double* fluxC, hiprandState *state, const int model_size) {
    const int thread_id = threadIdx.x+blockDim.x*blockIdx.x;
    const int i = thread_id + 1;
    if(i < model_size)
        fluxC[i] = hiprand_normal(&state[i]);
}

__global__
void Update(double* q2, double* q1, double *q0,hiprandState *state,  double* ct_c, double *fluxC,double *temperature_plot, const int n1_L, const int n1_R, const int n2_L, const int n2_R, const int n3_L, const int n3_R, const bool is_heat)
{
    const int thread_id = threadIdx.x+blockDim.x*blockIdx.x;
    const int i = thread_id + 1;
    if (i < n3_R) {
        for (int k = 0; k < statistics_interval; k++) {
            // 次の粒子状態の計算
            const int is_middleL = (n2_L < i && i <= n2_R);
            const int is_middleR = (n2_L <= i && i < n2_R);
            const double muL = is_middleL * mu1 + !(is_middleL) * mu1_bath;
            const double muR = is_middleR * mu1 + !(is_middleR) * mu1_bath;

            const double f_R = muR * (q1[i + 1] - q1[i]);
            const double f_L = muL * (q1[i] - q1[i - 1]);
            const double l_f = f_R - f_L;
            
            const double nf_R = (is_middleR) * beta0 * (q1[i + 1] - q1[i]) * (q1[i + 1] - q1[i]) * (q1[i + 1] - q1[i]);
            const double nf_L = (is_middleL) * beta0 * (q1[i - 1] - q1[i]) * (q1[i - 1] - q1[i]) * (q1[i - 1] - q1[i]);
            const double nl_f = nf_L + nf_R;

            const double dq1 = q1[i] - q0[i];
            const double random_f = (is_heat || i <= n1_R || i >= n3_L) ? (-gamma_t * dq1 + ct_c[i] * hiprand_normal(&state[i])) : 0;
            
            const double f = l_f + nl_f - mu0 * q1[i] + random_f;
            const double dq2 = dq1 + f * dt2 + random_f;  

            q2[i] = q1[i] + dq2;

            // 統計量の計算
            const double p = 0.5 * (q2[i] - q0[i]) * dt_1;
            fluxC[i] += p * (-f_L + nf_L);
            temperature_plot[i] += p * p;
            
            // __syncthreads();
            // if(thread_id == 0) {
            //     q0 = q1;
            //     q1 = q2;
            //     q2 = q0;
            // }
            // __syncthreads();
        }
    }
}

class FPUT_Lattice_1D{
private:
    int HeatBath_size;                      // 熱浴サイズ
    int middle_size;                        // 調べたいモデルサイズ
    int n1_L;                               // 高温熱浴の左端のインデックス
    int n1_R;                               // 高温熱浴の右端のインデックス
    int n2_L;                               // FPUT_Latticeの左端のインデックス
    int n2_R;                               // FPUT_Latticeの右端のインデックス
    int n3_L;                               // 低温熱浴の左端のインデックス
    int n3_R;                               // 低温熱浴の右端のインデックス
    int model_size;                         // モデル全体の粒子数

    long long int HeatSimulation;           // モデル全体に熱浴をかける（収束を早めたいため）
    long long int initialStateStep;         // 統計量を測定開始するステップ数（最初は収束していないため）
    long long int Step;                     // 全体のステップ数

    const double temp_h = 1.2;              // 高温熱浴の温度
    const double temp_l = 0.8;              // 低温熱浴の温度

    
    const double c_h = sqrt(2.0*gamma0*temp_h*dt);  // 高温側の分散
    const double c_l = sqrt(2.0*gamma0*temp_l*dt);  // 低温側の分散
    
    
    const double ct_h = c_h * dt;           
	const double ct_l = c_l * dt;        
    double* h_ct_c;           // 中央部分の（初期）温度  
    double* d_ct_c;           // 中央部分の（初期）温度    

    double* q0;                         // 2ステップ前の粒子位置
    double* q1;                         // 1ステップ前の粒子位置
    double* q2;                         // 現在の粒子位置    
    double* h_fluxC;
    double* h_temperature_plot;
    double* d_fluxC;
    double* d_temperature_plot;

    long long int count_flux = 0;
    long long int count_temp = 0;
    long long int stepCount = 0;

    hiprandState *state;   
public:
    
    FPUT_Lattice_1D() {};

    void settingSize(const int HeatBath_size_, const int middle_size_) {
        HeatBath_size = HeatBath_size_;
        middle_size = middle_size_;
        n1_L = 1;
        n1_R = n1_L + HeatBath_size - 1;
        n2_L = n1_R + 1;
        n2_R = n2_L + middle_size - 1;
        n3_L = n2_R + 1;
        n3_R = n3_L + HeatBath_size - 1;
        model_size = n3_R + 2;
        cerr << n1_L << "," << n1_R << "," << n2_L << "," << n2_R << "," << n3_L << "," << n3_R <<"," << model_size <<  endl;
        h_ct_c = (double *)malloc(sizeof(double) * model_size);
        h_fluxC = (double *)malloc(sizeof(double) * model_size);
        h_temperature_plot = (double *)malloc(sizeof(double) * model_size);
        hipMalloc(&d_ct_c, sizeof(double) * model_size);
        hipMalloc(&d_fluxC, sizeof(double) * model_size);
        hipMalloc(&d_temperature_plot, sizeof(double) * model_size);

        hipMalloc(&q0, sizeof(double) * model_size);
        hipMalloc(&q1, sizeof(double) * model_size);
        hipMalloc(&q2, sizeof(double) * model_size);
        hipMalloc(&state,  sizeof(double) * model_size * sizeof(hiprandState));
        for(int i = n1_L; i <= n1_R; i++) {
            h_ct_c[i] = ct_h;
        }
        for(int i = n2_L; i <= n2_R; i++) {
            const double temp_c = temp_h - (temp_h - temp_l) * (i - n2_L)/(n2_R - n2_L + 1);
            const double c_c = sqrt(2.0*gamma0*temp_c*dt); 
            h_ct_c[i] = c_c * dt;
        }
        for(int i = n3_L; i <= n3_R; i++) {
            h_ct_c[i] = ct_l;
        }
        hipMemcpy(d_ct_c, h_ct_c, sizeof(double) * model_size, hipMemcpyHostToDevice);
        dim3 grid_size = dim3(model_size / blockSize + 1, 1, 1);
        dim3 block_size = dim3(blockSize, 1, 1);
        setCurand<<<grid_size,block_size>>>(1, state);
        qinit<<<grid_size,block_size>>>(q0, q1, model_size);
    }
    void settingStep(const int HeatSimulation_, const int initialStateStep_, const int Step_) {
        HeatSimulation = HeatSimulation_;
        initialStateStep = initialStateStep_;
        Step = Step_;
    }
    // 進行度の出力
    void showProcessing() {
        // end = std::chrono::system_clock::now();
        // double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000.0 / 1000.0);
        // std::cerr << "process = " << std::fixed << std::setprecision(3) << (double)stepCount/(double)allSteps * 100.0 << "%, ";
        // std::cerr << "StepCount = " << std::setfill('0') << std::right << std::setw(9) << stepCount << "/" << std::setfill('0') << std::right << std::setw(9) << allSteps << ", ";
        // std::cerr << "t = " << std::fixed << std::setprecision(6) << time/86400.0 << " [days], ";
        // std::cerr << "est. = " << std::fixed << std::setprecision(6) << time/((double)stepCount/(double)allSteps)/86400.0<< " [days]" << std::endl;
        double *h_q0L = (double*) malloc(sizeof(double));
        double *h_q0M = (double*) malloc(sizeof(double));
        double *h_q0R = (double*) malloc(sizeof(double));
        hipMemcpy(h_q0L, q0 + (n1_L + n1_R) / 2, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_q0M, q0 + middle_size / 2, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_q0R, q0 + (n3_L + n3_R) / 2, sizeof(double), hipMemcpyDeviceToHost);
        std::cerr << *h_q0L << " " << *h_q0M << " " << *h_q0R << std::endl;
        assert(*h_q0M < 1e+10);

        free(h_q0L); free(h_q0M); free(h_q0R);
    }

    void step() {
        //void Update(double *q2, double* q1, double *q0, double ct_c, double *fluxC, curandState *state, double *temperature_plot, const int n1_L, const int n1_R, const int n2_L, const int n2_R, const int n3_L, const int n3_R, const bool is_heat, const double *f_nl)
        dim3 grid_size = dim3(model_size / blockSize + 1, 1, 1);
        dim3 block_size = dim3(blockSize, 1, 1);
        Update<<<grid_size,block_size>>>(q2, q1, q0, state,d_ct_c, d_fluxC, d_temperature_plot, n1_L, n1_R, n2_L, n2_R, n3_L, n3_R, stepCount < HeatSimulation);
        swap(q0, q1);
        swap(q1, q2);
        count_flux += statistics_interval;
        count_temp += statistics_interval;
        stepCount += statistics_interval;
    }
    void statistics_reset() {
        count_flux = 0;
        count_temp = 0;
        dim3 grid_size = dim3(model_size / blockSize + 1, 1, 1);
        dim3 block_size = dim3(blockSize, 1, 1);
        statistics_reset_d<<<grid_size,block_size>>>(d_fluxC, model_size);
        // for(int i = 0; i < model_size; i++) {
        //     temperature_plot[i] = 0;
        //     fluxC[i] = 0;
        // }
    }
    void output_Kappa() {
        dim3 grid_size = dim3(model_size / blockSize + 1, 1, 1);
        dim3 block_size = dim3(blockSize, 1, 1);
        //statistics_reset_test<<<grid_size,block_size>>>(d_fluxC, state, model_size);
        hipMemcpy(h_fluxC, d_fluxC, sizeof(double) * model_size, hipMemcpyDeviceToHost);
        double sum_flux = 0.0;
        int count0 = 0;
        for(int i = n2_L + 1; i < n2_R; i++) {
            sum_flux += h_fluxC[i];
            count0++;
        }
        const double ave_flux = sum_flux / count0 / count_flux;
        const double ave_kappa = ave_flux / (temp_h - temp_l) * middle_size;
        cerr << stepCount << "," << sum_flux << "," << ave_kappa << "," << count_flux << ",";

    }

    // void output_Temperature() {
    //     for(int i = 0; i < model_size; i++) {
    //         cerr << i << "," << temperature_plot[i] / count_temp << endl;
    //     }
    // }

};

int main(void) {
    FPUT_Lattice_1D model = FPUT_Lattice_1D();
    model.settingSize(20, 262144);
    model.settingStep(1000, 10000, 5000000000);
    std::chrono::system_clock::time_point  start, end; // 型は auto で可
    start = std::chrono::system_clock::now(); // 計測開始時間
 

    for(int i = 0; i < 5000000000/statistics_interval;i++) {
        if(i == 10000)
            model.statistics_reset();
        model.step();
        if((i + 1) % (500000/statistics_interval) == 0) {
            model.showProcessing();
            end = std::chrono::system_clock::now();  // 計測終了時間
            double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count(); //処理に要した時間をミリ秒に変換
            model.output_Kappa();
            cerr << elapsed/1000.0/60.0/60.0 << "[h]" << "," << elapsed/1000.0/60.0/60.0/i * 5000000000/statistics_interval << "[h]" << endl;
        }
    }
    //model.output_Temperature();
}